#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <thrust/random.h>
/*
 * Device code
 */
__device__
unsigned int hash(unsigned int a)
{
	a = (a+0x7ed55d16) + (a<<12);
	a = (a^0xc761c23c) ^ (a>>19);
	a = (a+0x165667b1) + (a<<5);
	a = (a+0xd3a2646c) ^ (a<<9);
	a = (a+0xfd7046c5) + (a<<3);
	a = (a^0xb55a4f09) ^ (a>>16);
	return a;
}
int __device__ gridmap(int const row, int const col,int const height)
{
	return row + col*height;
}
void __global__ mcmc(double const * const u,double const * const y,
                         double * const theta,
                         int const order,int const chain_length,int const threadsPerBlock)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
    int const globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    
    	int M = chain_length*threadsPerBlock;
    	if(globalIdx<M)
    	{	
    		unsigned int seed_normal = hash(globalIdx);
		thrust::default_random_engine rng_normal(seed_normal);
		thrust::random::experimental::normal_distribution<double> dist_norm(0, 1);
		
    		for(int ii=0;ii<chain_length;ii++)
    			theta[gridmap(chain_length*globalIdx+ii,1,M)] = dist_norm(rng_normal);
    	}	
    	
    
    
}

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *u_m;
    mxGPUArray const *y_m;
    double const *order_m;
    double const *chain_length_m;

    mxGPUArray *theta_m;
    
    double const *u;
    double const *y;
    double *theta;    
 

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 256;
    int blocksPerGrid;
    
    order_m =  (double *)mxGetData(prhs[2]); 
    int order = (int)*order_m;  
    mexPrintf("order = %d\n",order);
    
    chain_length_m =  (double *)mxGetData(prhs[3]); 
    int chain_length = (int)*chain_length_m;  
    mexPrintf("chain_length = %d\n",chain_length);

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    
    u_m = mxGPUCreateFromMxArray(prhs[0]);
    y_m = mxGPUCreateFromMxArray(prhs[1]);

    u = (double const *)(mxGPUGetDataReadOnly(u_m));
    y = (double const *)(mxGPUGetDataReadOnly(y_m));

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    mwSize const ndims = 2;
    mwSize const dim[] = {chain_length*threadsPerBlock,0,order,0};
    mxClassID const cid = mxDOUBLE_CLASS;
    mxComplexity const cxx = mxREAL;
    mxGPUInitialize const init0  =  MX_GPU_INITIALIZE_VALUES;    
    

    theta_m = mxGPUCreateGPUArray(ndims,dim,cid,cxx,init0);
    theta = (double *)(mxGPUGetData(theta_m));
    
    blocksPerGrid = 2;
    mcmc<<<blocksPerGrid, threadsPerBlock>>>(y,u,theta,order,chain_length,threadsPerBlock);
    
    
    

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(theta_m);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(y_m);
    mxGPUDestroyGPUArray(theta_m);
}
